#include "hip/hip_runtime.h"
/*
    Parameters
        transform: float**
            Wavelet変換した結果
        wavelets: float**
            スケーリングした後のWavelet
        waveform: float*
            変換の対象となる波形
*/
__global__ void wavelet_transform(float** transform, float** wavelets, float* waveform) {
    // threadIdx.x, blockDim.x
    int t = threadIdx.x;    // 時間軸
    int s = threadIdx.y;    // スケール軸

    // 畳み込み積分
    float total = 0;
    for (int i = 0; i < length; ++i) {
        total += waveforms[t] * wavelets[s][i];
    }
    transform[s][t] = total;
}

/*
有毛細胞の数は11,500個
記憶している長さ、44,100Hz(1秒)
計算量は480,000,000
グラボのCUDAコア1000個で割ると480,000
*/