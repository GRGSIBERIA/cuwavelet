#include "hip/hip_runtime.h"
/*
    @brief ウェーブレット変換を行う
    @param[out] transform Wavelet変換した結果
    @param[in] wavelets スケーリングした後のWavelet
    @param[in] waveform 変換の対象となる波形
    @param[in] timeN 時刻の長さ
    @param[in] scaleN スケールの数
*/
__global__ void wavelet_transform(
        float* transform, float* wavelets, float* waveform,
        int timeN, int scaleN
    ) {
    int t = blockDim.x * blockIdx.y + threadIdx.x;    // 時間軸
    int s = blockDim.y * blockIdx.y + threadIdx.y;    // スケール軸
    int idx = scaleN * s + t;

    // 畳み込み積分, f(t)g(i-t)の形式
    float total = 0;
    for (int i = t; i < timeN; ++i) {
        total += waveforms[t] * wavelets[s * scaleN + i - t];
    }
    transform[idx] = total;
}

/*
有毛細胞の数は11,500個
記憶している長さ、44,100Hz(1秒)
計算量は480,000,000
グラボのCUDAコア1000個で割ると480,000
*/